#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define TILE_WIDTH 32

void randomArray(int *A, int n, int k){
	srand((unsigned) time(NULL));
	for(int i=0; i< n*k; ++i)
		A[i] = ((int)rand()% 10) + 1;
	}

void printResults(int *h_matA, int *h_matB, int *h_matC, int n, int k, int m){
	printf("Matrix A:\n");
	for(int i=0; i< (n*k); i++){
		// int id = i + floor(i / (int)SQTILE_WIDTH )* (int)SQTILE_WIDTH;
		printf("%d	", h_matA[i]);
		if( (i+1) % k  == 0 ){
			printf("\n");
		}

	}
	printf("Matrix B:\n");
	for(int i=0; i< (k * m); i++){
		// int id = i + floor(i / (int)SQTILE_WIDTH )* (int)SQTILE_WIDTH;
		printf("%d	", h_matB[i]);
		if( (i+1) % m  == 0 ){
			printf("\n");
		}
	}

	printf("Matrix C:\n");
	for(int i=0; i< (n * m); i++){
		// int id = i + floor(i / (int)SQTILE_WIDTH )* (int)SQTILE_WIDTH;
		printf("%d	", h_matC[i]);
		if( (i+1) % m  == 0 ){
			printf("\n");
		}
	}
}

__global__ void matmul_rec_glob(int *a, int *b, int *c, int n, int k, int m) { 
    int row = blockIdx.y * blockDim.y + threadIdx.y; 
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if( col < m && row < n) {
    	int sum = 0;
        for(int i = 0; i < k; i++) {
            sum += a[row*k + i] * b[i*m + col];
        }
        c[row * m + col] = sum;
    }
}


__global__ void matmul_rec_shared(int *a, int *b, int *c, int n, int k, int m) {

	__shared__ int sA[TILE_WIDTH][TILE_WIDTH];
	__shared__ int sB[TILE_WIDTH][TILE_WIDTH];

    int bx = blockIdx.x, by = blockIdx.y,
       tx = threadIdx.x, ty = threadIdx.y,
       Row = by * TILE_WIDTH + ty,
       Col = bx * TILE_WIDTH + tx;
    int Pvalue = 0;

    for (int i = 0; i < (m-1)/TILE_WIDTH+1; ++i) {
       if (Row < n && i*TILE_WIDTH+tx < k)
          sA[ty][tx] = a[Row*k + i*TILE_WIDTH+tx];
       else
          sA[ty][tx] = 0;
        
       if (Col < m && i*TILE_WIDTH+ty < k)
          sB[ty][tx] = b[(i*TILE_WIDTH+ty)*m + Col];
       else
          sB[ty][tx] = 0;

       __syncthreads();
       for (int j = 0; j < TILE_WIDTH; ++j)
          Pvalue += sA[ty][j] * sB[j][tx];
       __syncthreads();
    }
    if (Row < n && Col < m)
       c[Row*m+Col] = Pvalue;
}

int main() {
	// Create matrices
	// A = nxk, B = kxm, C = nxm
	// printf("Enter valid dimension of matrices (A = nxk, B = kxm): \n");
	int n = 20000, k=20000, m=50000;
	// scanf("%d %d %d", &n, &k, &m);


	// Multiprocessing constants
	unsigned int grid_rows = ceil(n / TILE_WIDTH) < 1 ? 1 : ceil(n/TILE_WIDTH);
    unsigned int grid_cols = ceil(m / TILE_WIDTH) < 1 ? 1 : ceil(m/TILE_WIDTH);
	const dim3 threadsPerBlock(TILE_WIDTH,TILE_WIDTH); 	// Must not exceed 1024 (max thread per block)
	const dim3 blocksPerGrid( grid_cols, grid_rows);	

	// Initialize host matrices
	// For stream 1
	
	int *h_A1, *h_B1, *h_C1;
	clock_t h_alloctime = clock();
    hipHostMalloc((void **) &h_A1, sizeof(int)*k*n);
    hipHostMalloc((void **) &h_B1, sizeof(int)*m*k);
    hipHostMalloc((void **) &h_C1, sizeof(int)*m*n);
    printf("[**] Using tile width = %d...\n", TILE_WIDTH);
    printf("[**] Creating matrix A with dimension %d x %d...\n", n,k);
		randomArray(h_A1, n, k);
		printf("[**] Creating matrix B with dimension %d x %d...\n", k,m);
		randomArray(h_B1, k, m);
		printf("[**] CPU Allocation time for the matrices: %.6f sec \n",(double)(clock()-h_alloctime)/CLOCKS_PER_SEC );

	// Allocate memory space on the device 
    int *d_A1, *d_B1, *d_C1;
    clock_t d_alloctime = clock();
    hipMalloc((void **) &d_A1, sizeof(int)*k*n);
    hipMalloc((void **) &d_B1, sizeof(int)*m*k);
    hipMalloc((void **) &d_C1, sizeof(int)*n*m);
	hipMemcpy(d_A1, h_A1, sizeof(int)*k*n, hipMemcpyHostToDevice);
	hipMemcpy(d_B1, h_B1, sizeof(int)*k*m, hipMemcpyHostToDevice);
	printf("[**] GPU Allocation time for the matrices: %.6fsec \n",(double)(clock()-d_alloctime)/CLOCKS_PER_SEC );







	hipEvent_t start,end;
	float ms, avems = 0.0;



	printf("[**] Starting kernel program 'matmul_rec_glob' execution\n");
	for(int i = 0; i<10; i++) {
		hipEventCreate(&start);
		hipEventCreate(&end);
		hipEventRecord(start, 0);

		matmul_rec_glob<<< blocksPerGrid, threadsPerBlock >>>(d_A1, d_B1, d_C1, n, k, m);

		hipEventRecord(end, 0);
		hipEventSynchronize(end);
		hipEventElapsedTime(&ms, start, end);

		// printf("\tIteration no. %d: %.6fsecs\n", i, ms);
		avems+=ms;
		hipMemcpy(h_C, d_C, sizeof(int)*m*n, hipMemcpyDeviceToHost); 
		hipEventDestroy(start);
		hipEventDestroy(end);
	}
	printf("  >>> Average kernel execution time: %.6fsec.\n\n", avems/10.0);
	// printResults(h_A, h_B, h_C, n, k, m);

	printf("[**] Starting kernel program 'matmul_rec_shared' execution\n");
	






	avems = 0.0;

	
	for(int i = 0; i<10; i++) {
		hipEventCreate(&start);
		hipEventCreate(&end);
		hipEventRecord(start, 0);

		matmul_rec_shared<<< blocksPerGrid, threadsPerBlock >>>(d_A, d_B, d_C, n, k, m);

		hipEventRecord(end, 0);
		hipEventSynchronize(end);
		hipEventElapsedTime(&ms, start, end);

		// printf("\tIteration no. %d: %.6f sec\n", i, ms);
		avems+=ms;
		hipMemcpy(h_C, d_C, sizeof(int)*m*n, hipMemcpyDeviceToHost); 
		hipEventDestroy(start);
		hipEventDestroy(end);
	}
	printf("  >>> Average kernel execution time: %.6f sec.\n", avems/10.0);
	printf("[**] Freed memory. Done.\n\n");

	// printResults(h_A, h_B, h_C, n, k, m);

	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
	free(h_A);
	free(h_B);
	free(h_C);
	// printf("[**] Freed memory. Done.\n");
	return 0;
	
}